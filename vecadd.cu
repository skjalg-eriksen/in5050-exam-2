#define N 4
#define T 1024 // max threads per block

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>
__global__ void vecAdd (int *a, int *b, int *c);



int main() {
	uint8_t a[N] = {2, 4, 5, 1}; // 12
	uint8_t b[N] = {3, 7, 8, 1}; // 19
	int c[N];//=0;
	int *dev_a, *dev_b, *dev_c;
	// initialize a and b with real values (NOT SHOWN)
	int size = N * sizeof(int);
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	hipMemcpy(dev_a, a, size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size,hipMemcpyHostToDevice);
	
	vecAdd<<<1,4>>>(dev_a,dev_b,dev_c);
	printf("\naaa\n");
	hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);
	
	for (int i=0; i < sizeof(*c); i++){
		printf("vec %d\n", c[i]);		
	}
	//printf("vec %d\n", *c);		
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	//printf("\nvec %d\n", c);		
	
	exit (0);
}


__global__ void vecAdd (int *a, int *b, int *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N) {
		//c[0] += a[i] + b[i];
//		atomicAdd(&c[3], (int)(a[i]+b[i]));
		c[0] = __vsadu4(a[i], b[i]);
		
		printf("abs:%d\n", __vsadu4(a[i], b[i]));
	}


}

