#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

extern "C"{

#include "me.h"
}
#include "dsp.cuh"

/* Motion estimation for 8x8 block */
__global__ static void me_block_8x8(struct c63_common *cm,
    uint8_t *orig, uint8_t *ref, int color_component)
{
  int mb_y = blockIdx.x * blockDim.x + threadIdx.x;
  int mb_x = blockIdx.y * blockDim.y + threadIdx.y;

  struct macroblock *mb =
    &cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/8+mb_x];

  int range = cm->me_search_range;

  //Quarter resolution for chroma channels.
  if (color_component > 0) { range /= 2; }

  int left = mb_x * 8 - range;
  int top = mb_y * 8 - range;
  int right = mb_x * 8 + range;
  int bottom = mb_y * 8 + range;

  int w = cm->padw[color_component];
  int h = cm->padh[color_component];

  // Make sure we are within bounds of reference frame. TODO: Support partial frame bounds.
  if (left < 0) { left = 0; }
  if (top < 0) { top = 0; }
  if (right > (w - 8)) { right = w - 8; }
  if (bottom > (h - 8)) { bottom = h - 8; }

  int x, y;

  int mx = mb_x * 8;
  int my = mb_y * 8;

  int best_sad = INT_MAX;

    #pragma unroll
    for (y = top; y < bottom; ++y)
    {
      #pragma unroll
      for (x = left; x < right; ++x)
      {
          int sad = 0;

          uint8_t *block1 = orig + my*w+mx;
          uint8_t *block2 = ref + y*w+x;
          int stride = w;

            #pragma unroll
            for (int v = 0; v < 8; ++v)
            {
              #pragma unroll
              for (int u = 0; u < 8; ++u)
              {
                //*result += abs(block2[v*stride+u] - block1[v*stride+u]);
                  sad  += abs(block2[v*stride+u] - block1[v*stride+u]);
                  //__vsadu4
              }
            }


          if  (sad< best_sad)
          {
            mb->mv_x = x - mx;

            mb->mv_y = y - my;

            best_sad = sad;
          }


      }
    }



  /* Here, there should be a threshold on SAD that checks if the motion vector
     is cheaper than intraprediction. We always assume MV to be beneficial */

  //printf("(%d,%d) Using motion vector (%d, %d) with SAD %d\n",mb_x, mb_y, mb->mv_x, mb->mv_y, best_sad);

  mb->use_mv = 1;
}

/*
__global__ void test(int id){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  for (size_t i = 0; i < 1000; i++) {
    printf(" %d,", id);
  }
}*/

void c63_motion_estimate(struct c63_common *cm)
{
  //hipStream_t streams[3
  //hipStreamCreate(&streams[0]);
  //hipStreamCreate(&streams[1]);
  //hipStreamCreate(&streams[2]);

  hipStream_t y_stream, u_stream, v_stream;
  hipStreamCreate(&y_stream);
  hipStreamCreate(&u_stream);
  hipStreamCreate(&v_stream);
/*  hipEvent_t start;
  hipEventCreate(&start);

  hipStreamWaitEvent(y_stream, start,0);
  hipStreamWaitEvent(u_stream, start,0);
  hipStreamWaitEvent(v_stream, start,0);
  */
/*
  test<<<1,2,0,y_stream>>>(1);
  test<<<1,2,0,v_stream>>>(2);

  //hipStreamSynchronize(v_stream);
  //hipStreamSynchronize(y_stream);
  hipDeviceSynchronize();
  printf("\ndone\n");
  exit(1);*/
  /* Compare this frame with previous reconstructed frame */
  //int mb_x, mb_y;

  // <<<block_grid_UV, thread_grid>>>
  // block_grid_UV = (upw, uph)
  // thread_grid = (8,8)
  // Block grid: NUM_8x8BLOCKSxNUM_8x8BLOCKS U and V component

/*
  struct c63_common *y_cm;
  hipMalloc((void**)&y_cm, sizeof(struct c63_common));
  hipMemcpy(y_cm,cm, sizeof(cm),hipMemcpyHostToDevice);*/


  /* hipStreamAttachMemAsync(y_stream, cm);
  hipStreamAttachMemAsync(u_stream, cm);
  hipStreamAttachMemAsync(v_stream, cm);
 hipStreamAttachMemAsync(y_stream, cm->curframe->orig->Y);
  hipStreamAttachMemAsync(u_stream, cm->refframe->recons->Y);
  hipStreamAttachMemAsync(v_stream, Y_COMPONENT);*/

  //hipDeviceSynchronize();


  //hipMemcpy(cm, sizeof(cm), hipMemcpyHostToDevice, y_stream);
  dim3 Y_dim(cm->mb_rows, cm->mb_cols);

  me_block_8x8 <<<Y_dim, 1, 0 ,y_stream>>>(cm, cm->curframe->orig->Y,  cm->refframe->recons->Y, Y_COMPONENT);
  hipStreamSynchronize(y_stream);

  /* Chroma */

  dim3 UV_dim(cm->mb_rows / 2, cm->mb_cols / 2);
   me_block_8x8<<<UV_dim, 1, 0, u_stream>>> (cm, cm->curframe->orig->U,  cm->refframe->recons->U, U_COMPONENT);
  //hipDeviceSynchronize();
  hipStreamSynchronize(u_stream);

   me_block_8x8<<<UV_dim, 1, 0, v_stream>>> (cm, cm->curframe->orig->V,  cm->refframe->recons->V, V_COMPONENT);
   hipStreamSynchronize(v_stream);


/*
   hipEventRecord(start, y_stream);
   hipEventRecord(start, u_stream);
   hipEventRecord(start, v_stream);
*/
   //printf("sss wa?\n");
/*
  hipStreamSynchronize(u_stream);
  hipStreamSynchronize(v_stream);
  hipStreamSynchronize(y_stream);*/

  //hipDeviceReset();
  //printf("done V\n" );
  //hipDeviceSynchronize();
}

/* Motion compensation for 8x8 block */
__global__ static void mc_block_8x8(struct c63_common *cm,
    uint8_t *predicted, uint8_t *ref, int color_component)
{
  int mb_y = blockIdx.x * blockDim.x + threadIdx.x;
  int mb_x = blockIdx.y * blockDim.y + threadIdx.y;

  struct macroblock *mb =
    &cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/8+mb_x];

  if (!mb->use_mv) { return; }

  int left = mb_x * 8;
  int top = mb_y * 8;
  int right = left + 8;
  int bottom = top + 8;

  int w = cm->padw[color_component];

  /* Copy block from ref mandated by MV */
  int x, y;

  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      predicted[y*w+x] = ref[(y + mb->mv_y) * w + (x + mb->mv_x)];
    }
  }
}

void c63_motion_compensate(struct c63_common *cm)
{
  hipStream_t y_stream, u_stream, v_stream;
  hipStreamCreate(&y_stream);
  hipStreamCreate(&u_stream);
  hipStreamCreate(&v_stream);

  //int mb_x, mb_y;

  /* Luma */
  /*for (mb_y = 0; mb_y < cm->mb_rows; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols; ++mb_x)
    {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->Y,
          cm->refframe->recons->Y, Y_COMPONENT);
    }
  }*/
  dim3 Y_dim(cm->mb_rows, cm->mb_cols);
  dim3 UV_dim(cm->mb_rows / 2, cm->mb_cols / 2);

  mc_block_8x8 <<<Y_dim, 1, 0 ,y_stream>>> (cm, cm->curframe->predicted->Y, cm->refframe->recons->Y, Y_COMPONENT);
  hipStreamSynchronize(y_stream);
  /* Chroma */
  /*
  for (mb_y = 0; mb_y < cm->mb_rows / 2; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols / 2; ++mb_x)
    {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->U,
          cm->refframe->recons->U, U_COMPONENT);
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->V,
          cm->refframe->recons->V, V_COMPONENT);
    }
  }*/
  mc_block_8x8 <<<UV_dim, 1, 0, u_stream>>>  (cm, cm->curframe->predicted->U, cm->refframe->recons->U, U_COMPONENT);
  hipStreamSynchronize(u_stream);
  mc_block_8x8 <<<UV_dim, 1, 0, v_stream>>>  (cm, cm->curframe->predicted->V, cm->refframe->recons->V, V_COMPONENT);
  hipStreamSynchronize(v_stream);
}
